#include "hip/hip_runtime.h"
#include "ndt_gpu/MatrixHost.h"
#include "ndt_gpu/debug.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

namespace gpu {

MatrixHost::MatrixHost()
{
	fr_ = false;
}

MatrixHost::MatrixHost(int rows, int cols) {
	rows_ = rows;
	cols_ = cols;
	offset_ = 1;

	buffer_ = (double*)malloc(sizeof(double) * rows_ * cols_ * offset_);
	memset(buffer_, 0, sizeof(double) * rows_ * cols_ * offset_);
	fr_ = true;
}

MatrixHost::MatrixHost(int rows, int cols, int offset, double *buffer)
{
	rows_ = rows;
	cols_ = cols;
	offset_ = offset;
	buffer_ = buffer;
	fr_ = false;
}

MatrixHost::MatrixHost(const MatrixHost& other) {
	rows_ = other.rows_;
	cols_ = other.cols_;
	offset_ = other.offset_;
	fr_ = other.fr_;

	if (fr_) {
		buffer_ = (double*)malloc(sizeof(double) * rows_ * cols_ * offset_);
		memcpy(buffer_, other.buffer_, sizeof(double) * rows_ * cols_ * offset_);
	} else {
		buffer_ = other.buffer_;
	}
}

extern "C" __global__ void copyMatrixDevToDev(MatrixDevice input, MatrixDevice output) {
	int row = threadIdx.x;
	int col = threadIdx.y;
	int rows_num = input.rows();
	int cols_num = input.cols();

	if (row < rows_num && col < cols_num)
		output(row, col) = input(row, col);
}

bool MatrixHost::moveToGpu(MatrixDevice output) {
	if (rows_ != output.rows() || cols_ != output.cols())
		return false;

	if (offset_ == output.offset()) {
		checkCudaErrors(hipMemcpy(output.buffer(), buffer_, sizeof(double) * rows_ * cols_ * offset_, hipMemcpyHostToDevice));
		return true;
	}
	else {
		double *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(double) * rows_ * cols_ * offset_));
		checkCudaErrors(hipMemcpy(tmp, buffer_, sizeof(double) * rows_ * cols_ * offset_, hipMemcpyHostToDevice));

		MatrixDevice tmp_output(rows_, cols_, offset_, tmp);

		dim3 block_x(rows_, cols_, 1);
		dim3 grid_x(1, 1, 1);

		copyMatrixDevToDev<<<grid_x, block_x>>>(tmp_output, output);
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipFree(tmp));

		return true;
	}
}

bool MatrixHost::moveToHost(MatrixDevice input) {
	if (rows_ != input.rows() || cols_ != input.cols())
		return false;

	if (offset_ == input.offset()) {
		checkCudaErrors(hipMemcpy(buffer_, input.buffer(), sizeof(double) * rows_ * cols_ * offset_, hipMemcpyDeviceToHost));
		return true;
	}
	else {
		double *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(double) * rows_ * cols_ * offset_));

		MatrixDevice tmp_output(rows_, cols_, offset_, tmp);

		dim3 block_x(rows_, cols_, 1);
		dim3 grid_x(1, 1, 1);

		copyMatrixDevToDev << <grid_x, block_x >> >(input, tmp_output);
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipMemcpy(buffer_, tmp, sizeof(double) * rows_ * cols_ * offset_, hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(tmp));

		return true;
	}
}

MatrixHost &MatrixHost::operator=(const MatrixHost &other)
{
	rows_ = other.rows_;
	cols_ = other.cols_;
	offset_ = other.offset_;
	fr_ = other.fr_;

	if (fr_) {
		buffer_ = (double*)malloc(sizeof(double) * rows_ * cols_ * offset_);
		memcpy(buffer_, other.buffer_, sizeof(double) * rows_ * cols_ * offset_);
	} else {
		buffer_ = other.buffer_;
	}

	return *this;
}

void MatrixHost::debug()
{
	for (int i = 0; i < rows_; i++) {
		for (int j = 0; j < cols_; j++) {
			std::cout << buffer_[(i * cols_ + j) * offset_] << " ";
		}

		std::cout << std::endl;
	}

	std::cout << std::endl;
}

MatrixHost::~MatrixHost()
{
	if (fr_)
		free(buffer_);
}


SquareMatrixHost::SquareMatrixHost(int size) :
	 MatrixHost(size, size)
{

}

}
