#include "hip/hip_runtime.h"
#include "ndt_gpu/Registration.h"
#include "ndt_gpu/debug.h"
#include <iostream>

namespace gpu {

GRegistration::GRegistration()
{
	max_iterations_ = 0;
	x_ = y_ = z_ = NULL;
	points_number_ = 0;

	trans_x_ = trans_y_ = trans_z_ = NULL;

	converged_ = false;
	nr_iterations_ = 0;

	transformation_epsilon_ = 0;
	target_cloud_updated_ = true;
	target_points_number_ = 0;

	target_x_ = target_y_ = target_z_ = NULL;
	is_copied_ = false;

}

GRegistration::GRegistration(const GRegistration &other)
{
	transformation_epsilon_ = other.transformation_epsilon_;
	max_iterations_ = other.max_iterations_;

	//Original scanned point clouds
	x_ = other.x_;
	y_ = other.y_;
	z_ = other.z_;

	points_number_ = other.points_number_;

	trans_x_ = other.trans_x_;
	trans_y_ = other.trans_y_;
	trans_z_ = other.trans_z_;

	converged_ = other.converged_;

	nr_iterations_ = other.nr_iterations_;
	final_transformation_ = other.final_transformation_;
	transformation_ = other.transformation_;
	previous_transformation_ = other.previous_transformation_;

	target_cloud_updated_ = other.target_cloud_updated_;

	target_x_ = other.target_x_;
	target_y_ = other.target_y_;
	target_z_ = other.target_z_;

	target_points_number_ = other.target_points_number_;
	is_copied_ = true;
}

GRegistration::~GRegistration()
{
	if (!is_copied_) {
		if (x_ != NULL) {
			checkCudaErrors(hipFree(x_));
			x_ = NULL;
		}

		if (y_ != NULL) {
			checkCudaErrors(hipFree(y_));
			y_ = NULL;
		}

		if (z_ != NULL) {
			checkCudaErrors(hipFree(z_));
			z_ = NULL;
		}

		if (trans_x_ != NULL) {
			checkCudaErrors(hipFree(trans_x_));
			trans_x_ = NULL;
		}

		if (trans_y_ != NULL) {
			checkCudaErrors(hipFree(trans_y_));
			trans_y_ = NULL;
		}

		if (trans_z_ != NULL) {
			checkCudaErrors(hipFree(trans_z_));
			trans_z_ = NULL;
		}

		if (target_x_ != NULL) {
				checkCudaErrors(hipFree(target_x_));
			target_x_ = NULL;
		}

		if (target_y_ != NULL) {
			checkCudaErrors(hipFree(target_y_));
			target_y_ = NULL;
		}

		if (target_z_ != NULL) {
			checkCudaErrors(hipFree(target_z_));
			target_z_ = NULL;
		}
	}
}

void GRegistration::setTransformationEpsilon(double trans_eps)
{
	transformation_epsilon_ = trans_eps;
}

double GRegistration::getTransformationEpsilon() const
{
	return transformation_epsilon_;
}

void GRegistration::setMaximumIterations(int max_itr)
{
	max_iterations_ = max_itr;
}

int GRegistration::getMaximumIterations() const
{
	return max_iterations_;
}

Eigen::Matrix<float, 4, 4> GRegistration::getFinalTransformation() const
{
	return final_transformation_;
}

int GRegistration::getFinalNumIteration() const
{
	return nr_iterations_;
}

bool GRegistration::hasConverged() const
{
	return converged_;
}


template <typename T>
__global__ void convertInput(T *input, float *out_x, float *out_y, float *out_z, int point_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = idx; i < point_num; i += stride) {
		T tmp = input[i];
		out_x[i] = tmp.x;
		out_y[i] = tmp.y;
		out_z[i] = tmp.z;
	}
}

void GRegistration::setInputSource(pcl::PointCloud<pcl::PointXYZI>::Ptr input)
{
	//Convert point cloud to float x, y, z
	if (input->size() > 0) {
		points_number_ = input->size();

		pcl::PointXYZI *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZI) * points_number_));

		pcl::PointXYZI *host_tmp = input->points.data();

		// Pin the host buffer for accelerating the memory copy
#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZI) * points_number_, hipHostRegisterDefault));
#endif

		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZI) * points_number_, hipMemcpyHostToDevice));

		if (x_ != NULL) {
			checkCudaErrors(hipFree(x_));
			x_ = NULL;
		}

		if (y_ != NULL) {
			checkCudaErrors(hipFree(y_));
			y_ = NULL;
		}

		if (z_ != NULL) {
			checkCudaErrors(hipFree(z_));
			z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&z_, sizeof(float) * points_number_));

		int block_x = (points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : points_number_;
		int grid_x = (points_number_ - 1) / block_x + 1;

		convertInput<pcl::PointXYZI><<<grid_x, block_x>>>(tmp, x_, y_, z_, points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());


		if (trans_x_ != NULL) {
			checkCudaErrors(hipFree(trans_x_));
			trans_x_ = NULL;
		}

		if (trans_y_ != NULL) {
			checkCudaErrors(hipFree(trans_y_));
			trans_y_ = NULL;
		}

		if (trans_z_ != NULL) {
			checkCudaErrors(hipFree(trans_z_));
			trans_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&trans_x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_z_, sizeof(float) * points_number_));

		// Initially, also copy scanned points to transformed buffers
		checkCudaErrors(hipMemcpy(trans_x_, x_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(trans_y_, y_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(trans_z_, z_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));

		checkCudaErrors(hipFree(tmp));

		// Unpin host buffer
#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
	}
}

void GRegistration::setInputSource(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
	//Convert point cloud to float x, y, z
	if (input->size() > 0) {
		points_number_ = input->size();

		pcl::PointXYZ *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZ) * points_number_));

		pcl::PointXYZ *host_tmp = input->points.data();

		// Pin the host buffer for accelerating the memory copy
#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZ) * points_number_, hipHostRegisterDefault));
#endif

		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZ) * points_number_, hipMemcpyHostToDevice));

		if (x_ != NULL) {
			checkCudaErrors(hipFree(x_));
			x_ = NULL;
		}

		if (y_ != NULL) {
			checkCudaErrors(hipFree(y_));
			y_ = NULL;
		}

		if (z_ != NULL) {
			checkCudaErrors(hipFree(z_));
			z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&z_, sizeof(float) * points_number_));

		int block_x = (points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : points_number_;
		int grid_x = (points_number_ - 1) / block_x + 1;

		convertInput<pcl::PointXYZ><<<grid_x, block_x>>>(tmp, x_, y_, z_, points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		if (trans_x_ != NULL) {
			checkCudaErrors(hipFree(trans_x_));
			trans_x_ = NULL;
		}

		if (trans_y_ != NULL) {
			checkCudaErrors(hipFree(trans_y_));
			trans_y_ = NULL;
		}

		if (trans_z_ != NULL) {
			checkCudaErrors(hipFree(trans_z_));
			trans_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&trans_x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_z_, sizeof(float) * points_number_));

		checkCudaErrors(hipMemcpy(trans_x_, x_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(trans_y_, y_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(trans_z_, z_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));

		checkCudaErrors(hipFree(tmp));
#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
	}
}



//Set input MAP data
void GRegistration::setInputTarget(pcl::PointCloud<pcl::PointXYZI>::Ptr input)
{
	if (input->size() > 0) {
		target_points_number_ = input->size();

		pcl::PointXYZI *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZI) * target_points_number_));

		pcl::PointXYZI *host_tmp = input->points.data();

#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZI) * target_points_number_, hipHostRegisterDefault));
#endif

		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZI) * target_points_number_, hipMemcpyHostToDevice));

		if (target_x_ != NULL) {
			checkCudaErrors(hipFree(target_x_));
			target_x_ = NULL;
		}

		if (target_y_ != NULL) {
			checkCudaErrors(hipFree(target_y_));
			target_y_ = NULL;
		}

		if (target_z_ != NULL) {
			checkCudaErrors(hipFree(target_z_));
			target_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&target_x_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_y_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_z_, sizeof(float) * target_points_number_));

		int block_x = (target_points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : target_points_number_;
		int grid_x = (target_points_number_ - 1) / block_x + 1;

		convertInput<pcl::PointXYZI><<<grid_x, block_x>>>(tmp, target_x_, target_y_, target_z_, target_points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
		checkCudaErrors(hipFree(tmp));
	}
}

void GRegistration::setInputTarget(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
	if (input->size() > 0) {
		target_points_number_ = input->size();

		pcl::PointXYZ *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZ) * target_points_number_));

		pcl::PointXYZ *host_tmp = input->points.data();

#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZ) * target_points_number_, hipHostRegisterDefault));
#endif

		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZ) * target_points_number_, hipMemcpyHostToDevice));

		if (target_x_ != NULL) {
			checkCudaErrors(hipFree(target_x_));
			target_x_ = NULL;
		}

		if (target_y_ != NULL) {
			checkCudaErrors(hipFree(target_y_));
			target_y_ = NULL;
		}

		if (target_z_ != NULL) {
			checkCudaErrors(hipFree(target_z_));
			target_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&target_x_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_y_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_z_, sizeof(float) * target_points_number_));

		int block_x = (target_points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : target_points_number_;
		int grid_x = (target_points_number_ - 1) / block_x + 1;

		convertInput<pcl::PointXYZ><<<grid_x, block_x>>>(tmp, target_x_, target_y_, target_z_, target_points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipFree(tmp));
#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
	}
}

void GRegistration::align(const Eigen::Matrix<float, 4, 4> &guess)
{
	converged_ = false;

	final_transformation_ = transformation_ = previous_transformation_ = Eigen::Matrix<float, 4, 4>::Identity();

	computeTransformation(guess);
}

void GRegistration::computeTransformation(const Eigen::Matrix<float, 4, 4> &guess) {
	printf("Unsupported by Registration\n");
}

}
