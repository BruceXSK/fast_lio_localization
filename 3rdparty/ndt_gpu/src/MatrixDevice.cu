#include "ndt_gpu/MatrixDevice.h"
#include "ndt_gpu/debug.h"

namespace gpu {
MatrixDevice::MatrixDevice(int rows, int cols) {
	rows_ = rows;
	cols_ = cols;
	offset_ = 1;
	fr_ = true;
	buffer_ = NULL;
}

void MatrixDevice::memAlloc()
{
	if (buffer_ != NULL && fr_) {
		checkCudaErrors(hipFree(buffer_));
		buffer_ = NULL;
	}

	checkCudaErrors(hipMalloc(&buffer_, sizeof(double) * rows_ * cols_ * offset_));
	checkCudaErrors(hipMemset(buffer_, 0, sizeof(double) * rows_ * cols_ * offset_));
	checkCudaErrors(hipDeviceSynchronize());
	fr_ = true;
}

void MatrixDevice::memFree()
{
	if (fr_) {
		if (buffer_ != NULL) {
			checkCudaErrors(hipFree(buffer_));
			buffer_ = NULL;
		}
	}
}


SquareMatrixDevice::SquareMatrixDevice(int size) :
	MatrixDevice(size, size)
{

}

}
